#include "Thrust_interop.hpp"
#include <opencv2/core/cuda_stream_accessor.hpp>

#include <thrust/transform.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
//! [prg]
struct prg
{
  float a, b;

  __host__ __device__
    prg(float _a = 0.f, float _b = 1.f) : a(_a), b(_b) {};

  __host__ __device__
    float operator()(const unsigned int n) const
  {
    thrust::default_random_engine rng;
    thrust::uniform_real_distribution<float> dist(a, b);
    rng.discard(n);
    return dist(rng);
  }
};
//! [prg]


//! [pred_greater]
template<typename T> struct pred_greater
{
  T value;
  __host__ __device__ pred_greater(T value_) : value(value_){}
  __host__ __device__ bool operator()(const T& val) const
  {
    return val > value;
  }
};
//! [pred_greater]


int main(void)
{
  // Generate a 2 channel row matrix with 100 elements.  Set the first channel to be the element index, and the second to be a randomly
  // generated value.  Sort by the randomly generated value while maintaining index association.
  //! [sort]
  {
    cv::cuda::GpuMat d_data(1, 100, CV_32SC2);
    // Thrust compatible begin and end iterators to channel 1 of this matrix
    auto keyBegin = GpuMatBeginItr<int>(d_data, 1);
    auto keyEnd = GpuMatEndItr<int>(d_data, 1);
    // Thrust compatible begin and end iterators to channel 0 of this matrix
    auto idxBegin = GpuMatBeginItr<int>(d_data, 0);
    auto idxEnd = GpuMatEndItr<int>(d_data, 0);
    // Fill the index channel with a sequence of numbers from 0 to 100
    thrust::sequence(idxBegin, idxEnd);
    // Fill the key channel with random numbers between 0 and 10.  A counting iterator is used here to give an integer value for each location as an input to prg::operator()
    thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(d_data.cols), keyBegin, prg(0, 10));
    // Sort the key channel and index channel such that the keys and indices stay together
    thrust::sort_by_key(keyBegin, keyEnd, idxBegin);

    cv::Mat h_idx(d_data);
  }
  //! [sort]

  // Randomly fill a row matrix with 100 elements between -1 and 1
  //! [random]
  {
    cv::cuda::GpuMat d_value(1, 100, CV_32F);
    auto valueBegin = GpuMatBeginItr<float>(d_value);
    auto valueEnd = GpuMatEndItr<float>(d_value);
    thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(d_value.cols), valueBegin, prg(-1, 1));

    cv::Mat h_value(d_value);
  }
  //! [random]

  // OpenCV has count non zero, but what if you want to count a specific value?
  //! [count_value]
  {
    cv::cuda::GpuMat d_value(1, 100, CV_32S);
    d_value.setTo(cv::Scalar(0));
    d_value.colRange(10, 50).setTo(cv::Scalar(15));
    auto count = thrust::count(GpuMatBeginItr<int>(d_value), GpuMatEndItr<int>(d_value), 15);
    std::cout << count << std::endl;
  }
  //! [count_value]

  // Randomly fill an array then copy only values greater than 0.  Perform these tasks on a stream.
  //! [copy_greater]
  {
    cv::cuda::GpuMat d_value(1, 100, CV_32F);
    auto valueBegin = GpuMatBeginItr<float>(d_value);
    auto valueEnd = GpuMatEndItr<float>(d_value);
    cv::cuda::Stream stream;
    //! [random_gen_stream]
    // Same as the random generation code from before except now the transformation is being performed on a stream
    thrust::transform(thrust::system::cuda::par.on(cv::cuda::StreamAccessor::getStream(stream)), thrust::make_counting_iterator(0), thrust::make_counting_iterator(d_value.cols), valueBegin, prg(-1, 1));
    //! [random_gen_stream]
    // Count the number of values we are going to copy
    int count = thrust::count_if(thrust::system::cuda::par.on(cv::cuda::StreamAccessor::getStream(stream)), valueBegin, valueEnd, pred_greater<float>(0.0));
    // Allocate a destination for copied values
    cv::cuda::GpuMat d_valueGreater(1, count, CV_32F);
    // Copy values that satisfy the predicate.
    thrust::copy_if(thrust::system::cuda::par.on(cv::cuda::StreamAccessor::getStream(stream)), valueBegin, valueEnd, GpuMatBeginItr<float>(d_valueGreater), pred_greater<float>(0.0));
    cv::Mat h_greater(d_valueGreater);
  }
  //! [copy_greater]

  return 0;
}
