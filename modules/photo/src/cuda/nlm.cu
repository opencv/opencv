#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"

using namespace cv::cuda;

typedef unsigned char uchar;
typedef unsigned short ushort;

//////////////////////////////////////////////////////////////////////////////////
//// Non Local Means Denosing

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        __device__ __forceinline__ float norm2(const float& v) { return v*v; }
        __device__ __forceinline__ float norm2(const float2& v) { return v.x*v.x + v.y*v.y; }
        __device__ __forceinline__ float norm2(const float3& v) { return v.x*v.x + v.y*v.y + v.z*v.z; }
        __device__ __forceinline__ float norm2(const float4& v) { return v.x*v.x + v.y*v.y + v.z*v.z  + v.w*v.w; }

        template<typename T, typename B>
        __global__ void nlm_kernel(const PtrStep<T> src, PtrStepSz<T> dst, const B b, int search_radius, int block_radius, float noise_mult)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

            const int i = blockDim.y * blockIdx.y + threadIdx.y;
            const int j = blockDim.x * blockIdx.x + threadIdx.x;

            if (j >= dst.cols || i >= dst.rows)
                return;

            int bsize = search_radius + block_radius;
            int search_window = 2 * search_radius + 1;
            float minus_search_window2_inv = -1.f/(search_window * search_window);

            value_type sum1 = VecTraits<value_type>::all(0);
            float sum2 = 0.f;

            if (j - bsize >= 0 && j + bsize < dst.cols && i - bsize >= 0 && i + bsize < dst.rows)
            {
                for(float y = -search_radius; y <= search_radius; ++y)
                    for(float x = -search_radius; x <= search_radius; ++x)
                    {
                        float dist2 = 0;
                        for(float ty = -block_radius; ty <= block_radius; ++ty)
                            for(float tx = -block_radius; tx <= block_radius; ++tx)
                            {
                                value_type bv = saturate_cast<value_type>(src(i + y + ty, j + x + tx));
                                value_type av = saturate_cast<value_type>(src(i +     ty, j +     tx));

                                dist2 += norm2(av - bv);
                            }

                        float w = __expf(dist2 * noise_mult + (x * x + y * y) * minus_search_window2_inv);

                        /*if (i == 255 && j == 255)
                            printf("%f %f\n", w, dist2 * minus_h2_inv + (x * x + y * y) * minus_search_window2_inv);*/

                        sum1 = sum1 + w * saturate_cast<value_type>(src(i + y, j + x));
                        sum2 += w;
                    }
            }
            else
            {
                for(float y = -search_radius; y <= search_radius; ++y)
                    for(float x = -search_radius; x <= search_radius; ++x)
                    {
                        float dist2 = 0;
                        for(float ty = -block_radius; ty <= block_radius; ++ty)
                            for(float tx = -block_radius; tx <= block_radius; ++tx)
                            {
                                value_type bv = saturate_cast<value_type>(b.at(i + y + ty, j + x + tx, src));
                                value_type av = saturate_cast<value_type>(b.at(i +     ty, j +     tx, src));
                                dist2 += norm2(av - bv);
                            }

                        float w = __expf(dist2 * noise_mult + (x * x + y * y) * minus_search_window2_inv);

                        sum1 = sum1 + w * saturate_cast<value_type>(b.at(i + y, j + x, src));
                        sum2 += w;
                    }

            }

            dst(i, j) = saturate_cast<T>(sum1 / sum2);

        }

        template<typename T, template <typename> class B>
        void nlm_caller(const PtrStepSzb src, PtrStepSzb dst, int search_radius, int block_radius, float h, hipStream_t stream)
        {
            dim3 block (32, 8);
            dim3 grid (divUp (src.cols, block.x), divUp (src.rows, block.y));

            B<T> b(src.rows, src.cols);

            int block_window = 2 * block_radius + 1;
            float minus_h2_inv = -1.f/(h * h * VecTraits<T>::cn);
            float noise_mult = minus_h2_inv/(block_window * block_window);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(nlm_kernel<T), B<T> >, hipFuncCachePreferL1) );
            nlm_kernel<<<grid, block>>>((PtrStepSz<T>)src, (PtrStepSz<T>)dst, b, search_radius, block_radius, noise_mult);
            cudaSafeCall ( hipGetLastError () );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template<typename T>
        void nlm_bruteforce_gpu(const PtrStepSzb& src, PtrStepSzb dst, int search_radius, int block_radius, float h, int borderMode, hipStream_t stream)
        {
            typedef void (*func_t)(const PtrStepSzb src, PtrStepSzb dst, int search_radius, int block_radius, float h, hipStream_t stream);

            static func_t funcs[] =
            {
                nlm_caller<T, BrdConstant>,
                nlm_caller<T, BrdReplicate>,
                nlm_caller<T, BrdReflect>,
                nlm_caller<T, BrdWrap>,
                nlm_caller<T, BrdReflect101>
            };
            funcs[borderMode](src, dst, search_radius, block_radius, h, stream);
        }

        template void nlm_bruteforce_gpu<uchar>(const PtrStepSzb&, PtrStepSzb, int, int, float, int, hipStream_t);
        template void nlm_bruteforce_gpu<uchar2>(const PtrStepSzb&, PtrStepSzb, int, int, float, int, hipStream_t);
        template void nlm_bruteforce_gpu<uchar3>(const PtrStepSzb&, PtrStepSzb, int, int, float, int, hipStream_t);

        template void nlm_bruteforce_gpu<ushort>(const PtrStepSzb&, PtrStepSzb, int, int, float, int, hipStream_t);
    }
}}}

//////////////////////////////////////////////////////////////////////////////////
//// Non Local Means Denosing (fast approximate version)

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {

        template <int cn> struct Unroll;
        template <> struct Unroll<1>
        {
            template <int BLOCK_SIZE>
            static __device__ __forceinline__ thrust::tuple<volatile float*, volatile float*> smem_tuple(float* smem)
            {
                return cv::cuda::device::smem_tuple(smem, smem + BLOCK_SIZE);
            }

            static __device__ __forceinline__ thrust::tuple<float&, float&> tie(float& val1, float& val2)
            {
                return thrust::tie(val1, val2);
            }

            static __device__ __forceinline__ const thrust::tuple<plus<float>, plus<float> > op()
            {
                plus<float> op;
                return thrust::make_tuple(op, op);
            }
        };
        template <> struct Unroll<2>
        {
            template <int BLOCK_SIZE>
            static __device__ __forceinline__ thrust::tuple<volatile float*, volatile float*, volatile float*> smem_tuple(float* smem)
            {
                return cv::cuda::device::smem_tuple(smem, smem + BLOCK_SIZE, smem + 2 * BLOCK_SIZE);
            }

            static __device__ __forceinline__ thrust::tuple<float&, float&, float&> tie(float& val1, float2& val2)
            {
                return thrust::tie(val1, val2.x, val2.y);
            }

            static __device__ __forceinline__ const thrust::tuple<plus<float>, plus<float>, plus<float> > op()
            {
                plus<float> op;
                return thrust::make_tuple(op, op, op);
            }
        };
        template <> struct Unroll<3>
        {
            template <int BLOCK_SIZE>
            static __device__ __forceinline__ thrust::tuple<volatile float*, volatile float*, volatile float*, volatile float*> smem_tuple(float* smem)
            {
                return cv::cuda::device::smem_tuple(smem, smem + BLOCK_SIZE, smem + 2 * BLOCK_SIZE, smem + 3 * BLOCK_SIZE);
            }

            static __device__ __forceinline__ thrust::tuple<float&, float&, float&, float&> tie(float& val1, float3& val2)
            {
                return thrust::tie(val1, val2.x, val2.y, val2.z);
            }

            static __device__ __forceinline__ const thrust::tuple<plus<float>, plus<float>, plus<float>, plus<float> > op()
            {
                plus<float> op;
                return thrust::make_tuple(op, op, op, op);
            }
        };
        template <> struct Unroll<4>
        {
            template <int BLOCK_SIZE>
            static __device__ __forceinline__ thrust::tuple<volatile float*, volatile float*, volatile float*, volatile float*, volatile float*> smem_tuple(float* smem)
            {
                return cv::cuda::device::smem_tuple(smem, smem + BLOCK_SIZE, smem + 2 * BLOCK_SIZE, smem + 3 * BLOCK_SIZE, smem + 4 * BLOCK_SIZE);
            }

            static __device__ __forceinline__ thrust::tuple<float&, float&, float&, float&, float&> tie(float& val1, float4& val2)
            {
                return thrust::tie(val1, val2.x, val2.y, val2.z, val2.w);
            }

            static __device__ __forceinline__ const thrust::tuple<plus<float>, plus<float>, plus<float>, plus<float>, plus<float> > op()
            {
                plus<float> op;
                return thrust::make_tuple(op, op, op, op, op);
            }
        };

        __device__ __forceinline__ int calcDist(const uchar&  a, const uchar&  b) { return (a-b)*(a-b); }
        __device__ __forceinline__ int calcDist(const ushort& a, const ushort& b) { return ::abs(a - b);}
        __device__ __forceinline__ int calcDist(const uchar2& a, const uchar2& b) { return (a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y); }
        __device__ __forceinline__ int calcDist(const uchar3& a, const uchar3& b) { return (a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y) + (a.z-b.z)*(a.z-b.z); }

        template <class T> struct FastNonLocalMeans
        {
            enum
            {
                CTA_SIZE = 128,

                TILE_COLS = 128,
                TILE_ROWS = 32,

                STRIDE = CTA_SIZE
            };

            struct plus
            {
                __device__ __forceinline__ float operator()(float v1, float v2) const { return v1 + v2; }
            };

            int search_radius;
            int block_radius;

            int search_window;
            int block_window;
            float minus_h2_inv;

            FastNonLocalMeans(int search_window_, int block_window_, float h) : search_radius(search_window_/2), block_radius(block_window_/2),
                search_window(search_window_), block_window(block_window_), minus_h2_inv(-1.f/(h * h * VecTraits<T>::cn)) {}

            PtrStep<T> src;
            mutable PtrStepi buffer;

            __device__ __forceinline__ void initSums_BruteForce(int i, int j, int* dist_sums, PtrStepi& col_sums, PtrStepi& up_col_sums) const
            {
                for(int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
                {
                    dist_sums[index] = 0;

                    for(int tx = 0; tx < block_window; ++tx)
                        col_sums(tx, index) = 0;

                    int y = index / search_window;
                    int x = index - y * search_window;

                    int ay = i;
                    int ax = j;

                    int by = i + y - search_radius;
                    int bx = j + x - search_radius;

#if 1
                    for (int tx = -block_radius; tx <= block_radius; ++tx)
                    {
                        int col_sum = 0;
                        for (int ty = -block_radius; ty <= block_radius; ++ty)
                        {
                            int dist = calcDist(src(ay + ty, ax + tx), src(by + ty, bx + tx));

                            dist_sums[index] += dist;
                            col_sum += dist;
                        }
                        col_sums(tx + block_radius, index) = col_sum;
                    }
#else
                    for (int ty = -block_radius; ty <= block_radius; ++ty)
                        for (int tx = -block_radius; tx <= block_radius; ++tx)
                        {
                            int dist = calcDist(src(ay + ty, ax + tx), src(by + ty, bx + tx));

                            dist_sums[index] += dist;
                            col_sums(tx + block_radius, index) += dist;
                        }
#endif

                    up_col_sums(j, index) = col_sums(block_window - 1, index);
                }
            }

            __device__ __forceinline__ void shiftRight_FirstRow(int i, int j, int first, int* dist_sums, PtrStepi& col_sums, PtrStepi& up_col_sums) const
            {
                for(int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
                {
                    int y = index / search_window;
                    int x = index - y * search_window;

                    int ay = i;
                    int ax = j + block_radius;

                    int by = i + y - search_radius;
                    int bx = j + x - search_radius + block_radius;

                    int col_sum = 0;

                    for (int ty = -block_radius; ty <= block_radius; ++ty)
                        col_sum += calcDist(src(ay + ty, ax), src(by + ty, bx));

                    dist_sums[index] += col_sum - col_sums(first, index);

                    col_sums(first, index) = col_sum;
                    up_col_sums(j, index) = col_sum;
                }
            }

            __device__ __forceinline__ void shiftRight_UpSums(int i, int j, int first, int* dist_sums, PtrStepi& col_sums, PtrStepi& up_col_sums) const
            {
                int ay = i;
                int ax = j + block_radius;

                T a_up   = src(ay - block_radius - 1, ax);
                T a_down = src(ay + block_radius, ax);

                for(int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
                {
                    int y = index / search_window;
                    int x = index - y * search_window;

                    int by = i + y - search_radius;
                    int bx = j + x - search_radius + block_radius;

                    T b_up   = src(by - block_radius - 1, bx);
                    T b_down = src(by + block_radius, bx);

                    int col_sum = up_col_sums(j, index) + calcDist(a_down, b_down) - calcDist(a_up, b_up);

                    dist_sums[index] += col_sum  - col_sums(first, index);
                    col_sums(first, index) = col_sum;
                    up_col_sums(j, index) = col_sum;
                }
            }

            __device__ __forceinline__ void convolve_window(int i, int j, const int* dist_sums, T& dst) const
            {
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type sum_type;

                float weights_sum = 0;
                sum_type sum = VecTraits<sum_type>::all(0);

                float bw2_inv = 1.f/(block_window * block_window);

                int sx = j - search_radius;
                int sy = i - search_radius;

                for(int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
                {
                    int y = index / search_window;
                    int x = index - y * search_window;

                    float avg_dist = dist_sums[index] * bw2_inv;
                    float weight = __expf(avg_dist * minus_h2_inv);
                    weights_sum += weight;

                    sum = sum + weight * saturate_cast<sum_type>(src(sy + y, sx + x));
                }

                __shared__ float cta_buffer[CTA_SIZE * (VecTraits<T>::cn + 1)];

                reduce<CTA_SIZE>(Unroll<VecTraits<T>::cn>::template smem_tuple<CTA_SIZE>(cta_buffer),
                                 Unroll<VecTraits<T>::cn>::tie(weights_sum, sum),
                                 threadIdx.x,
                                 Unroll<VecTraits<T>::cn>::op());

                if (threadIdx.x == 0)
                    dst = saturate_cast<T>(sum / weights_sum);
            }

            __device__ __forceinline__ void operator()(PtrStepSz<T>& dst) const
            {
                int tbx = blockIdx.x * TILE_COLS;
                int tby = blockIdx.y * TILE_ROWS;

                int tex = ::min(tbx + TILE_COLS, dst.cols);
                int tey = ::min(tby + TILE_ROWS, dst.rows);

                PtrStepi col_sums;
                col_sums.data = buffer.ptr(dst.cols + blockIdx.x * block_window) + blockIdx.y * search_window * search_window;
                col_sums.step = buffer.step;

                PtrStepi up_col_sums;
                up_col_sums.data = buffer.data + blockIdx.y * search_window * search_window;
                up_col_sums.step = buffer.step;

                extern __shared__ int dist_sums[]; //search_window * search_window

                int first = 0;

                for (int i = tby; i < tey; ++i)
                    for (int j = tbx; j < tex; ++j)
                    {
                        __syncthreads();

                        if (j == tbx)
                        {
                            initSums_BruteForce(i, j, dist_sums, col_sums, up_col_sums);
                            first = 0;
                        }
                        else
                        {
                            if (i == tby)
                              shiftRight_FirstRow(i, j, first, dist_sums, col_sums, up_col_sums);
                            else
                              shiftRight_UpSums(i, j, first, dist_sums, col_sums, up_col_sums);

                            first = (first + 1) % block_window;
                        }

                        __syncthreads();

                        convolve_window(i, j, dist_sums, dst(i, j));
                    }
            }

        };

        template<typename T>
        __global__ void fast_nlm_kernel(const FastNonLocalMeans<T> fnlm, PtrStepSz<T> dst) { fnlm(dst); }

        void nln_fast_get_buffer_size(const PtrStepSzb& src, int search_window, int block_window, int& buffer_cols, int& buffer_rows)
        {
            typedef FastNonLocalMeans<uchar> FNLM;
            dim3 grid(divUp(src.cols, FNLM::TILE_COLS), divUp(src.rows, FNLM::TILE_ROWS));

            buffer_cols = search_window * search_window * grid.y;
            buffer_rows = src.cols + block_window * grid.x;
        }

        template<typename T>
        void nlm_fast_gpu(const PtrStepSzb& src, PtrStepSzb dst, PtrStepi buffer,
                          int search_window, int block_window, float h, hipStream_t stream)
        {
            typedef FastNonLocalMeans<T> FNLM;
            FNLM fnlm(search_window, block_window, h);

            fnlm.src = (PtrStepSz<T>)src;
            fnlm.buffer = buffer;

            dim3 block(FNLM::CTA_SIZE, 1);
            dim3 grid(divUp(src.cols, FNLM::TILE_COLS), divUp(src.rows, FNLM::TILE_ROWS));
            int smem = search_window * search_window * sizeof(int);


            fast_nlm_kernel<<<grid, block, smem>>>(fnlm, (PtrStepSz<T>)dst);
            cudaSafeCall ( hipGetLastError () );
            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void nlm_fast_gpu<uchar>(const PtrStepSzb&, PtrStepSzb, PtrStepi, int, int, float,  hipStream_t);
        template void nlm_fast_gpu<uchar2>(const PtrStepSzb&, PtrStepSzb, PtrStepi, int, int, float, hipStream_t);
        template void nlm_fast_gpu<uchar3>(const PtrStepSzb&, PtrStepSzb, PtrStepi, int, int, float, hipStream_t);
        template void nlm_fast_gpu<ushort>(const PtrStepSzb&, PtrStepSzb, PtrStepi, int, int, float, hipStream_t);

        __global__ void fnlm_split_kernel(const PtrStepSz<uchar3> lab, PtrStepb l, PtrStep<uchar2> ab)
        {
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;

            if (x < lab.cols && y < lab.rows)
            {
                uchar3 p = lab(y, x);
                ab(y,x) = make_uchar2(p.y, p.z);
                l(y,x) = p.x;
            }
        }

        void fnlm_split_channels(const PtrStepSz<uchar3>& lab, PtrStepb l, PtrStep<uchar2> ab, hipStream_t stream)
        {
            dim3 b(32, 8);
            dim3 g(divUp(lab.cols, b.x), divUp(lab.rows, b.y));

            fnlm_split_kernel<<<g, b>>>(lab, l, ab);
            cudaSafeCall ( hipGetLastError () );
            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void fnlm_merge_kernel(const PtrStepb l, const PtrStep<uchar2> ab, PtrStepSz<uchar3> lab)
        {
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;

            if (x < lab.cols && y < lab.rows)
            {
                uchar2 p = ab(y, x);
                lab(y, x) = make_uchar3(l(y, x), p.x, p.y);
            }
        }

        void fnlm_merge_channels(const PtrStepb& l, const PtrStep<uchar2>& ab, PtrStepSz<uchar3> lab, hipStream_t stream)
        {
            dim3 b(32, 8);
            dim3 g(divUp(lab.cols, b.x), divUp(lab.rows, b.y));

            fnlm_merge_kernel<<<g, b>>>(l, ab, lab);
            cudaSafeCall ( hipGetLastError () );
            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}
