#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/core/cuda.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/cuda/utility.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

device::ThrustAllocator::~ThrustAllocator()
{
}
namespace
{
    class DefaultThrustAllocator: public cv::cuda::device::ThrustAllocator
    {
    public:
        __device__ __host__ uchar* allocate(size_t numBytes) CV_OVERRIDE
        {
#ifndef __CUDA_ARCH__
            uchar* ptr;
            CV_CUDEV_SAFE_CALL(hipMalloc(&ptr, numBytes));
            return ptr;
#else
            return NULL;
#endif
        }
        __device__ __host__ void deallocate(uchar* ptr, size_t numBytes) CV_OVERRIDE
        {
            CV_UNUSED(numBytes);
#ifndef __CUDA_ARCH__
            CV_CUDEV_SAFE_CALL(hipFree(ptr));
#endif
        }
    };
    DefaultThrustAllocator defaultThrustAllocator;
    cv::cuda::device::ThrustAllocator* g_thrustAllocator = &defaultThrustAllocator;
}


cv::cuda::device::ThrustAllocator& cv::cuda::device::ThrustAllocator::getAllocator()
{
    return *g_thrustAllocator;
}

void cv::cuda::device::ThrustAllocator::setAllocator(cv::cuda::device::ThrustAllocator* allocator)
{
    if(allocator == NULL)
        g_thrustAllocator = &defaultThrustAllocator;
    else
        g_thrustAllocator = allocator;
}

namespace
{
    class DefaultAllocator : public GpuMat::Allocator
    {
    public:
        bool allocate(GpuMat* mat, int rows, int cols, size_t elemSize) CV_OVERRIDE;
        void free(GpuMat* mat) CV_OVERRIDE;
    };

    bool DefaultAllocator::allocate(GpuMat* mat, int rows, int cols, size_t elemSize)
    {
        if (rows > 1 && cols > 1)
        {
            CV_CUDEV_SAFE_CALL( hipMallocPitch(&mat->data, &mat->step, elemSize * cols, rows) );
        }
        else
        {
            // Single row or single column must be continuous
            CV_CUDEV_SAFE_CALL( hipMalloc(&mat->data, elemSize * cols * rows) );
            mat->step = elemSize * cols;
        }

        mat->refcount = (int*) fastMalloc(sizeof(int));

        return true;
    }

    void DefaultAllocator::free(GpuMat* mat)
    {
        hipFree(mat->datastart);
        fastFree(mat->refcount);
    }

    DefaultAllocator cudaDefaultAllocator;
    GpuMat::Allocator* g_defaultAllocator = &cudaDefaultAllocator;
    GpuMat::Allocator* g_stdAllocator = &cudaDefaultAllocator;
}

GpuMat::Allocator* cv::cuda::GpuMat::defaultAllocator()
{
    return g_defaultAllocator;
}

void cv::cuda::GpuMat::setDefaultAllocator(Allocator* allocator)
{
    CV_Assert( allocator != 0 );
    g_defaultAllocator = allocator;
}

GpuMat::Allocator* cv::cuda::GpuMat::getStdAllocator()
{
    return g_stdAllocator;
}


/////////////////////////////////////////////////////
/// create

void cv::cuda::GpuMat::create(int _rows, int _cols, int _type)
{
    CV_DbgAssert( _rows >= 0 && _cols >= 0 );

    _type &= Mat::TYPE_MASK;

    if (rows == _rows && cols == _cols && type() == _type && data)
        return;

    if (data)
        release();

    if (_rows > 0 && _cols > 0)
    {
        flags = Mat::MAGIC_VAL + _type;
        rows = _rows;
        cols = _cols;

        const size_t esz = elemSize();

        bool allocSuccess = allocator->allocate(this, rows, cols, esz);

        if (!allocSuccess)
        {
            // custom allocator fails, try default allocator
            allocator = defaultAllocator();
            allocSuccess = allocator->allocate(this, rows, cols, esz);
            CV_Assert( allocSuccess );
        }

        if (esz * cols == step)
            flags |= Mat::CONTINUOUS_FLAG;

        datastart = data;
        dataend = data + step * (rows - 1) + cols * esz;

        if (refcount)
            *refcount = 1;
    }
}

/////////////////////////////////////////////////////
/// release

void cv::cuda::GpuMat::release()
{
    CV_DbgAssert( allocator != 0 );

    if (refcount && CV_XADD(refcount, -1) == 1)
        allocator->free(this);

    dataend = data = datastart = 0;
    step = rows = cols = 0;
    refcount = 0;
}

/////////////////////////////////////////////////////
/// upload

void cv::cuda::GpuMat::upload(InputArray arr)
{
    Mat mat = arr.getMat();

    CV_DbgAssert( !mat.empty() );

    create(mat.size(), mat.type());

    CV_CUDEV_SAFE_CALL( hipMemcpy2D(data, step, mat.data, mat.step, cols * elemSize(), rows, hipMemcpyHostToDevice) );
}

void cv::cuda::GpuMat::upload(InputArray arr, Stream& _stream)
{
    Mat mat = arr.getMat();

    CV_DbgAssert( !mat.empty() );

    create(mat.size(), mat.type());

    hipStream_t stream = StreamAccessor::getStream(_stream);
    CV_CUDEV_SAFE_CALL( hipMemcpy2DAsync(data, step, mat.data, mat.step, cols * elemSize(), rows, hipMemcpyHostToDevice, stream) );
}

/////////////////////////////////////////////////////
/// download

void cv::cuda::GpuMat::download(OutputArray _dst) const
{
    CV_DbgAssert( !empty() );

    _dst.create(size(), type());
    Mat dst = _dst.getMat();

    CV_CUDEV_SAFE_CALL( hipMemcpy2D(dst.data, dst.step, data, step, cols * elemSize(), rows, hipMemcpyDeviceToHost) );
}

void cv::cuda::GpuMat::download(OutputArray _dst, Stream& _stream) const
{
    CV_DbgAssert( !empty() );

    _dst.create(size(), type());
    Mat dst = _dst.getMat();

    hipStream_t stream = StreamAccessor::getStream(_stream);
    CV_CUDEV_SAFE_CALL( hipMemcpy2DAsync(dst.data, dst.step, data, step, cols * elemSize(), rows, hipMemcpyDeviceToHost, stream) );
}

/////////////////////////////////////////////////////
/// copyTo

void cv::cuda::GpuMat::copyTo(OutputArray _dst) const
{
    CV_DbgAssert( !empty() );

    _dst.create(size(), type());
    GpuMat dst = _dst.getGpuMat();

    CV_CUDEV_SAFE_CALL( hipMemcpy2D(dst.data, dst.step, data, step, cols * elemSize(), rows, hipMemcpyDeviceToDevice) );
}

void cv::cuda::GpuMat::copyTo(OutputArray _dst, Stream& _stream) const
{
    CV_DbgAssert( !empty() );

    _dst.create(size(), type());
    GpuMat dst = _dst.getGpuMat();

    hipStream_t stream = StreamAccessor::getStream(_stream);
    CV_CUDEV_SAFE_CALL( hipMemcpy2DAsync(dst.data, dst.step, data, step, cols * elemSize(), rows, hipMemcpyDeviceToDevice, stream) );
}

namespace
{
    template <size_t size> struct CopyToPolicy : DefaultTransformPolicy
    {
    };
    template <> struct CopyToPolicy<4> : DefaultTransformPolicy
    {
        enum {
            shift = 2
        };
    };
    template <> struct CopyToPolicy<8> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename T>
    void copyWithMask(const GpuMat& src, const GpuMat& dst, const GpuMat& mask, Stream& stream)
    {
        gridTransformUnary_< CopyToPolicy<sizeof(typename VecTraits<T>::elem_type)> >(globPtr<T>(src), globPtr<T>(dst), identity<T>(), globPtr<uchar>(mask), stream);
    }
}

void cv::cuda::GpuMat::copyTo(OutputArray _dst, InputArray _mask, Stream& stream) const
{
    CV_DbgAssert( !empty() );
    CV_DbgAssert( depth() <= CV_64F && channels() <= 4 );

    GpuMat mask = _mask.getGpuMat();
    CV_DbgAssert( size() == mask.size() && mask.depth() == CV_8U && (mask.channels() == 1 || mask.channels() == channels()) );

    uchar* data0 = _dst.getGpuMat().data;

    _dst.create(size(), type());
    GpuMat dst = _dst.getGpuMat();

    // do not leave dst uninitialized
    if (dst.data != data0)
        dst.setTo(Scalar::all(0), stream);

    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, const GpuMat& mask, Stream& stream);
    static const func_t funcs[9][4] =
    {
        {0,0,0,0},
        {copyWithMask<uchar>, copyWithMask<uchar2>, copyWithMask<uchar3>, copyWithMask<uchar4>},
        {copyWithMask<ushort>, copyWithMask<ushort2>, copyWithMask<ushort3>, copyWithMask<ushort4>},
        {0,0,0,0},
        {copyWithMask<int>, copyWithMask<int2>, copyWithMask<int3>, copyWithMask<int4>},
        {0,0,0,0},
        {0,0,0,0},
        {0,0,0,0},
        {copyWithMask<double>, copyWithMask<double2>, copyWithMask<double3>, copyWithMask<double4>}
    };

    if (mask.channels() == channels())
    {
        const func_t func = funcs[elemSize1()][0];
        CV_DbgAssert( func != 0 );
        func(reshape(1), dst.reshape(1), mask.reshape(1), stream);
    }
    else
    {
        const func_t func = funcs[elemSize1()][channels() - 1];
        CV_DbgAssert( func != 0 );
        func(*this, dst, mask, stream);
    }
}

/////////////////////////////////////////////////////
/// setTo

namespace
{
    template <typename T>
    void setToWithOutMask(const GpuMat& mat, Scalar _scalar, Stream& stream)
    {
        Scalar_<typename VecTraits<T>::elem_type> scalar = _scalar;
        gridTransformUnary(constantPtr(VecTraits<T>::make(scalar.val), mat.rows, mat.cols), globPtr<T>(mat), identity<T>(), stream);
    }

    template <typename T>
    void setToWithMask(const GpuMat& mat, const GpuMat& mask, Scalar _scalar, Stream& stream)
    {
        Scalar_<typename VecTraits<T>::elem_type> scalar = _scalar;
        gridTransformUnary(constantPtr(VecTraits<T>::make(scalar.val), mat.rows, mat.cols), globPtr<T>(mat), identity<T>(), globPtr<uchar>(mask), stream);
    }
}

GpuMat& cv::cuda::GpuMat::setTo(Scalar value, Stream& stream)
{
    CV_DbgAssert( !empty() );
    CV_DbgAssert( depth() <= CV_64F && channels() <= 4 );

    if (value[0] == 0.0 && value[1] == 0.0 && value[2] == 0.0 && value[3] == 0.0)
    {
        // Zero fill

        if (stream)
            CV_CUDEV_SAFE_CALL( hipMemset2DAsync(data, step, 0, cols * elemSize(), rows, StreamAccessor::getStream(stream)) );
        else
            CV_CUDEV_SAFE_CALL( hipMemset2D(data, step, 0, cols * elemSize(), rows) );

        return *this;
    }

    if (depth() == CV_8U)
    {
        const int cn = channels();

        if (cn == 1
                || (cn == 2 && value[0] == value[1])
                || (cn == 3 && value[0] == value[1] && value[0] == value[2])
                || (cn == 4 && value[0] == value[1] && value[0] == value[2] && value[0] == value[3]))
        {
            const int val = cv::saturate_cast<uchar>(value[0]);

            if (stream)
                CV_CUDEV_SAFE_CALL( hipMemset2DAsync(data, step, val, cols * elemSize(), rows, StreamAccessor::getStream(stream)) );
            else
                CV_CUDEV_SAFE_CALL( hipMemset2D(data, step, val, cols * elemSize(), rows) );

            return *this;
        }
    }

    typedef void (*func_t)(const GpuMat& mat, Scalar scalar, Stream& stream);
    static const func_t funcs[7][4] =
    {
        {setToWithOutMask<uchar>,setToWithOutMask<uchar2>,setToWithOutMask<uchar3>,setToWithOutMask<uchar4>},
        {setToWithOutMask<schar>,setToWithOutMask<char2>,setToWithOutMask<char3>,setToWithOutMask<char4>},
        {setToWithOutMask<ushort>,setToWithOutMask<ushort2>,setToWithOutMask<ushort3>,setToWithOutMask<ushort4>},
        {setToWithOutMask<short>,setToWithOutMask<short2>,setToWithOutMask<short3>,setToWithOutMask<short4>},
        {setToWithOutMask<int>,setToWithOutMask<int2>,setToWithOutMask<int3>,setToWithOutMask<int4>},
        {setToWithOutMask<float>,setToWithOutMask<float2>,setToWithOutMask<float3>,setToWithOutMask<float4>},
        {setToWithOutMask<double>,setToWithOutMask<double2>,setToWithOutMask<double3>,setToWithOutMask<double4>}
    };

    funcs[depth()][channels() - 1](*this, value, stream);

    return *this;
}

GpuMat& cv::cuda::GpuMat::setTo(Scalar value, InputArray _mask, Stream& stream)
{
    CV_DbgAssert( !empty() );
    CV_DbgAssert( depth() <= CV_64F && channels() <= 4 );

    GpuMat mask = _mask.getGpuMat();

    if (mask.empty())
    {
        return setTo(value, stream);
    }

    CV_DbgAssert( size() == mask.size() && mask.type() == CV_8UC1 );

    typedef void (*func_t)(const GpuMat& mat, const GpuMat& mask, Scalar scalar, Stream& stream);
    static const func_t funcs[7][4] =
    {
        {setToWithMask<uchar>,setToWithMask<uchar2>,setToWithMask<uchar3>,setToWithMask<uchar4>},
        {setToWithMask<schar>,setToWithMask<char2>,setToWithMask<char3>,setToWithMask<char4>},
        {setToWithMask<ushort>,setToWithMask<ushort2>,setToWithMask<ushort3>,setToWithMask<ushort4>},
        {setToWithMask<short>,setToWithMask<short2>,setToWithMask<short3>,setToWithMask<short4>},
        {setToWithMask<int>,setToWithMask<int2>,setToWithMask<int3>,setToWithMask<int4>},
        {setToWithMask<float>,setToWithMask<float2>,setToWithMask<float3>,setToWithMask<float4>},
        {setToWithMask<double>,setToWithMask<double2>,setToWithMask<double3>,setToWithMask<double4>}
    };

    funcs[depth()][channels() - 1](*this, mask, value, stream);

    return *this;
}

/////////////////////////////////////////////////////
/// convertTo

namespace
{
    template <typename T> struct ConvertToPolicy : DefaultTransformPolicy
    {
    };
    template <> struct ConvertToPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename T, typename D>
    void convertToNoScale(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        typedef typename VecTraits<T>::elem_type src_elem_type;
        typedef typename VecTraits<D>::elem_type dst_elem_type;
        typedef typename LargerType<src_elem_type, float>::type larger_elem_type;
        typedef typename LargerType<float, dst_elem_type>::type scalar_type;

        gridTransformUnary_< ConvertToPolicy<scalar_type> >(globPtr<T>(src), globPtr<D>(dst), saturate_cast_func<T, D>(), stream);
    }

    template <typename T, typename D, typename S> struct Convertor : unary_function<T, D>
    {
        S alpha;
        S beta;

        __device__ __forceinline__ D operator ()(typename TypeTraits<T>::parameter_type src) const
        {
            return cudev::saturate_cast<D>(alpha * src + beta);
        }
    };

    template <typename T, typename D>
    void convertToScale(const GpuMat& src, const GpuMat& dst, double alpha, double beta, Stream& stream)
    {
        typedef typename VecTraits<T>::elem_type src_elem_type;
        typedef typename VecTraits<D>::elem_type dst_elem_type;
        typedef typename LargerType<src_elem_type, float>::type larger_elem_type;
        typedef typename LargerType<float, dst_elem_type>::type scalar_type;

        Convertor<T, D, scalar_type> op;
        op.alpha = cv::saturate_cast<scalar_type>(alpha);
        op.beta = cv::saturate_cast<scalar_type>(beta);

        gridTransformUnary_< ConvertToPolicy<scalar_type> >(globPtr<T>(src), globPtr<D>(dst), op, stream);
    }

    template <typename T, typename D>
    void convertScaleHalf(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        typedef typename VecTraits<T>::elem_type src_elem_type;
        typedef typename VecTraits<D>::elem_type dst_elem_type;
        typedef typename LargerType<src_elem_type, float>::type larger_elem_type;
        typedef typename LargerType<float, dst_elem_type>::type scalar_type;

        gridTransformUnary_< ConvertToPolicy<scalar_type> >(globPtr<T>(src), globPtr<D>(dst), saturate_cast_fp16_func<T,D>(), stream);
    }
}

void cv::cuda::GpuMat::convertTo(OutputArray _dst, int rtype, Stream& stream) const
{
    if (rtype < 0)
        rtype = type();
    else
        rtype = CV_MAKE_TYPE(CV_MAT_DEPTH(rtype), channels());

    const int sdepth = depth();
    const int ddepth = CV_MAT_DEPTH(rtype);
    if (sdepth == ddepth)
    {
        if (stream)
            copyTo(_dst, stream);
        else
            copyTo(_dst);

        return;
    }

    CV_DbgAssert( sdepth <= CV_64F && ddepth <= CV_64F );

    GpuMat src = *this;

    _dst.create(size(), rtype);
    GpuMat dst = _dst.getGpuMat();

    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[7][7] =
    {
        {0, convertToNoScale<uchar, schar>, convertToNoScale<uchar, ushort>, convertToNoScale<uchar, short>, convertToNoScale<uchar, int>, convertToNoScale<uchar, float>, convertToNoScale<uchar, double>},
        {convertToNoScale<schar, uchar>, 0, convertToNoScale<schar, ushort>, convertToNoScale<schar, short>, convertToNoScale<schar, int>, convertToNoScale<schar, float>, convertToNoScale<schar, double>},
        {convertToNoScale<ushort, uchar>, convertToNoScale<ushort, schar>, 0, convertToNoScale<ushort, short>, convertToNoScale<ushort, int>, convertToNoScale<ushort, float>, convertToNoScale<ushort, double>},
        {convertToNoScale<short, uchar>, convertToNoScale<short, schar>, convertToNoScale<short, ushort>, 0, convertToNoScale<short, int>, convertToNoScale<short, float>, convertToNoScale<short, double>},
        {convertToNoScale<int, uchar>, convertToNoScale<int, schar>, convertToNoScale<int, ushort>, convertToNoScale<int, short>, 0, convertToNoScale<int, float>, convertToNoScale<int, double>},
        {convertToNoScale<float, uchar>, convertToNoScale<float, schar>, convertToNoScale<float, ushort>, convertToNoScale<float, short>, convertToNoScale<float, int>, 0, convertToNoScale<float, double>},
        {convertToNoScale<double, uchar>, convertToNoScale<double, schar>, convertToNoScale<double, ushort>, convertToNoScale<double, short>, convertToNoScale<double, int>, convertToNoScale<double, float>, 0}
    };

    funcs[sdepth][ddepth](src.reshape(1), dst.reshape(1), stream);
}

void cv::cuda::GpuMat::convertTo(OutputArray _dst, int rtype, double alpha, double beta, Stream& stream) const
{
    if (rtype < 0)
        rtype = type();
    else
        rtype = CV_MAKETYPE(CV_MAT_DEPTH(rtype), channels());

    const int sdepth = depth();
    const int ddepth = CV_MAT_DEPTH(rtype);

    GpuMat src = *this;

    _dst.create(size(), rtype);
    GpuMat dst = _dst.getGpuMat();

    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, double alpha, double beta, Stream& stream);
    static const func_t funcs[7][7] =
    {
        {convertToScale<uchar, uchar>, convertToScale<uchar, schar>, convertToScale<uchar, ushort>, convertToScale<uchar, short>, convertToScale<uchar, int>, convertToScale<uchar, float>, convertToScale<uchar, double>},
        {convertToScale<schar, uchar>, convertToScale<schar, schar>, convertToScale<schar, ushort>, convertToScale<schar, short>, convertToScale<schar, int>, convertToScale<schar, float>, convertToScale<schar, double>},
        {convertToScale<ushort, uchar>, convertToScale<ushort, schar>, convertToScale<ushort, ushort>, convertToScale<ushort, short>, convertToScale<ushort, int>, convertToScale<ushort, float>, convertToScale<ushort, double>},
        {convertToScale<short, uchar>, convertToScale<short, schar>, convertToScale<short, ushort>, convertToScale<short, short>, convertToScale<short, int>, convertToScale<short, float>, convertToScale<short, double>},
        {convertToScale<int, uchar>, convertToScale<int, schar>, convertToScale<int, ushort>, convertToScale<int, short>, convertToScale<int, int>, convertToScale<int, float>, convertToScale<int, double>},
        {convertToScale<float, uchar>, convertToScale<float, schar>, convertToScale<float, ushort>, convertToScale<float, short>, convertToScale<float, int>, convertToScale<float, float>, convertToScale<float, double>},
        {convertToScale<double, uchar>, convertToScale<double, schar>, convertToScale<double, ushort>, convertToScale<double, short>, convertToScale<double, int>, convertToScale<double, float>, convertToScale<double, double>}
    };

    funcs[sdepth][ddepth](src.reshape(1), dst.reshape(1), alpha, beta, stream);
}

void cv::cuda::convertFp16(InputArray _src, OutputArray _dst, Stream& stream)
{
    GpuMat src = _src.getGpuMat();
    int ddepth = 0;

    switch(src.depth())
    {
    case CV_32F:
        ddepth = CV_16S;
        break;
    case CV_16S:
        ddepth = CV_32F;
        break;
    default:
        CV_Error(Error::StsUnsupportedFormat, "Unsupported input depth");
        return;
    }
    int type = CV_MAKE_TYPE(CV_MAT_DEPTH(ddepth), src.channels());
    _dst.create(src.size(), type);
    GpuMat dst = _dst.getGpuMat();

    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        0, 0, 0,
        convertScaleHalf<float, short>, 0, convertScaleHalf<short, float>,
        0, 0,
    };

    funcs[ddepth](src.reshape(1), dst.reshape(1), stream);
}

#endif
