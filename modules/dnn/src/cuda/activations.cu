#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "functors.hpp"
#include "types.hpp"
#include "vector_traits.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include "../cuda4dnn/kernels/scale_shift.hpp"

#include <opencv2/core.hpp>

#include <cstddef>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn  { namespace kernels {

namespace raw {
    template <class T, class ActivationOp, std::size_t N>
    __global__ void generic_op_vec(Span<T> output, View<T> input, const typename ActivationOp::Params params) {
        using vector_type = get_vector_type_t<T, N>;

        auto output_vPtr = vector_type::get_pointer(output.data());
        auto input_vPtr = vector_type::get_pointer(input.data());

        ActivationOp activation_op(params);

        for (auto i : grid_stride_range(output.size() / vector_type::size())) {
            vector_type vec;
            v_load(vec, input_vPtr[i]);
            for (int j = 0; j < vector_type::size(); j++)
                vec.data[j] = activation_op(vec.data[j]);
            v_store(output_vPtr[i], vec);
        }
    }

    template <class T, std::size_t N>
    __global__ void axiswise_relu_vec(Span<T> output, View<T> input, size_type inner_size, View<T> slope) {
        using vector_type = get_vector_type_t<T, N>;

        auto output_vPtr = vector_type::get_pointer(output.data());
        auto input_vPtr = vector_type::get_pointer(input.data());

        for (auto i : grid_stride_range(output.size() / vector_type::size())) {
            const index_type c = (i / inner_size) % slope.size();

            vector_type vec;
            v_load(vec, input_vPtr[i]);
            for (int j = 0; j < vector_type::size(); j++)
                vec.data[j] = vec.data[j] > T(0) ? vec.data[j] : vec.data[j] * slope[c];
            v_store(output_vPtr[i], vec);
        }
    }

} /* namespace raw */

template <class T, class ActivationOp, std::size_t N> static
void launch_vectorized_generic_op(const Stream& stream, Span<T> output, View<T> input, const typename ActivationOp::Params& params) {
    CV_Assert(is_fully_aligned<T>(output, N));
    CV_Assert(is_fully_aligned<T>(input, N));

    auto kernel = raw::generic_op_vec<T, ActivationOp, N>;
    auto policy = make_policy(kernel, output.size() / N, 0, stream);
    launch_kernel(kernel, policy, output, input, params);
}

template <class T, class ActivationOp> static
void generic_op(const Stream& stream, Span<T> output, View<T> input, const typename ActivationOp::Params& params = {}) {
    CV_Assert(input.size() == output.size());

    if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(input, 4)) {
        launch_vectorized_generic_op<T, ActivationOp, 4>(stream, output, input, params);
    } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(input, 2)) {
        launch_vectorized_generic_op<T, ActivationOp, 2>(stream, output, input, params);
    } else {
        launch_vectorized_generic_op<T, ActivationOp, 1>(stream, output, input, params);
    }
}

template <class T>
void relu(const Stream& stream, Span<T> output, View<T> input, T slope) {
    generic_op<T, ReLUFunctor<T>>(stream, output, input, {slope});
}

template <class T>
void clipped_relu(const Stream& stream, Span<T> output, View<T> input, T floor, T ceiling) {
    CV_Assert(static_cast<double>(floor) <= static_cast<double>(ceiling));
    generic_op<T, ClippedReLUFunctor<T>>(stream, output, input, {floor, ceiling});
}

template <class T>
void tanh(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, TanHFunctor<T>>(stream, output, input);
}

template <class T>
void swish(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SwishFunctor<T>>(stream, output, input);
}

template <class T>
void mish(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, MishFunctor<T>>(stream, output, input);
}

template <class T>
void sigmoid(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SigmoidFunctor<T>>(stream, output, input);
}

template <class T>
void elu(const Stream& stream, Span<T> output, View<T> input, T alpha) {
    generic_op<T, ELUFunctor<T>>(stream, output, input, {alpha});
}

template <class T>
void bnll(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, BNLLFunctor<T>>(stream, output, input);
}

template <class T>
void ceil(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, CeilFunctor<T>>(stream, output, input);
}

template <class T>
void floor(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, FloorFunctor<T>>(stream, output, input);
}

template <class T>
void log(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, LogFunctor<T>>(stream, output, input);
}

template <class T>
void rint(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, RintFunctor<T>>(stream, output, input);
}

template <class T>
void sqrt(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SqrtFunctor<T>>(stream, output, input);
}

template <class T>
void not_k(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, NotFunctor<T>>(stream, output, input);
}

template <class T>
void acos(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AcosFunctor<T>>(stream, output, input);
}

template <class T>
void acosh(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AcoshFunctor<T>>(stream, output, input);
}

template <class T>
void asin(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AsinFunctor<T>>(stream, output, input);
}

template <class T>
void asinh(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AsinhFunctor<T>>(stream, output, input);
}

template <class T>
void atan(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AtanFunctor<T>>(stream, output, input);
}

template <class T>
void atanh(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AtanhFunctor<T>>(stream, output, input);
}

template <class T>
void cos(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, CosFunctor<T>>(stream, output, input);
}

template <class T>
void cosh(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, CoshFunctor<T>>(stream, output, input);
}

template <class T>
void erf(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, ErfFunctor<T>>(stream, output, input);
}

template <class T>
void hardswish(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, HardSwishFunctor<T>>(stream, output, input);
}

template <class T>
void sin(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SinFunctor<T>>(stream, output, input);
}

template <class T>
void sinh(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SinhFunctor<T>>(stream, output, input);
}

template <class T>
void softplus(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SoftplusFunctor<T>>(stream, output, input);
}

template <class T>
void softsign(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SoftsignFunctor<T>>(stream, output, input);
}

template <class T>
void tan(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, TanFunctor<T>>(stream, output, input);
}

template <class T>
void celu(const Stream& stream, Span<T> output, View<T> input, T alpha) {
    generic_op<T, CeluFunctor<T>>(stream, output, input, {alpha});
}

template <class T>
void hardsigmoid(const Stream& stream, Span<T> output, View<T> input, T alpha, T beta) {
    generic_op<T, HardSigmoidFunctor<T>>(stream, output, input, {alpha, beta});
}

template <class T>
void selu(const Stream& stream, Span<T> output, View<T> input, T alpha, T gamma) {
    generic_op<T, SeluFunctor<T>>(stream, output, input, {alpha, gamma});
}

template <class T>
void gelu(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, GeluFunctor<T>>(stream, output, input);
}

template <class T>
void sign(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, SignFunctor<T>>(stream, output, input);
}

template <class T>
void shrink(const Stream& stream, Span<T> output, View<T> input, T bias, T lambd) {
    generic_op<T, ShrinkFunctor<T>>(stream, output, input, {bias, lambd});
}

template <class T>
void reciprocal(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, ReciprocalFunctor<T>>(stream, output, input);
}

template <class T>
void thresholdedrelu(const Stream& stream, Span<T> output, View<T> input, T alpha) {
    generic_op<T, ThresholdedReluFunctor<T>>(stream, output, input, {alpha});
}

template <class T>
void abs(const Stream& stream, Span<T> output, View<T> input) {
    generic_op<T, AbsFunctor<T>>(stream, output, input);
}

template <class T>
void power(const Stream& stream, Span<T> output, View<T> input, T exp, T scale, T shift) {
    CV_Assert(input.size() == output.size());

    if (static_cast<float>(exp) == 1.0f) {
        scale1_with_bias1(stream, output, input, scale, shift);
        return;
    }

    generic_op<T, PowerFunctor<T>>(stream, output, input, {exp, scale, shift});
}

template <class T>
void exp(const Stream& stream, Span<T> output, View<T> input, T normScale, T normShift) {
    generic_op<T, ExpFunctor<T>>(stream, output, input, {normScale, normShift});
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void relu<__half>(const Stream&, Span<__half>, View<__half>, __half);
template void clipped_relu<__half>(const Stream&, Span<__half>, View<__half>, __half, __half);
template void tanh<__half>(const Stream&, Span<__half>, View<__half>);
template void swish<__half>(const Stream&, Span<__half>, View<__half>);
template void mish<__half>(const Stream&, Span<__half>, View<__half>);
template void sigmoid<__half>(const Stream&, Span<__half>, View<__half>);
template void elu<__half>(const Stream&, Span<__half>, View<__half>, __half);
template void abs<__half>(const Stream& stream, Span<__half> output, View<__half> input);
template void bnll<__half>(const Stream&, Span<__half>, View<__half>);
template void ceil<__half>(const Stream&, Span<__half>, View<__half>);
template void floor<__half>(const Stream&, Span<__half>, View<__half>);
template void log<__half>(const Stream&, Span<__half>, View<__half>);
template void rint<__half>(const Stream&, Span<__half>, View<__half>);
template void sqrt<__half>(const Stream&, Span<__half>, View<__half>);
template void not_k<__half>(const Stream&, Span<__half>, View<__half>);
template void acos<__half>(const Stream&, Span<__half>, View<__half>);
template void acosh<__half>(const Stream&, Span<__half>, View<__half>);
template void asin<__half>(const Stream&, Span<__half>, View<__half>);
template void asinh<__half>(const Stream&, Span<__half>, View<__half>);
template void atan<__half>(const Stream&, Span<__half>, View<__half>);
template void atanh<__half>(const Stream&, Span<__half>, View<__half>);
template void cos<__half>(const Stream&, Span<__half>, View<__half>);
template void cosh<__half>(const Stream&, Span<__half>, View<__half>);
template void erf<__half>(const Stream&, Span<__half>, View<__half>);
template void hardswish<__half>(const Stream&, Span<__half>, View<__half>);
template void sin<__half>(const Stream&, Span<__half>, View<__half>);
template void sinh<__half>(const Stream&, Span<__half>, View<__half>);
template void softplus<__half>(const Stream&, Span<__half>, View<__half>);
template void softsign<__half>(const Stream&, Span<__half>, View<__half>);
template void tan<__half>(const Stream&, Span<__half>, View<__half>);
template void celu<__half>(const Stream&, Span<__half>, View<__half>, __half);
template void hardsigmoid<__half>(const Stream&, Span<__half>, View<__half>, __half, __half);
template void selu<__half>(const Stream&, Span<__half>, View<__half>, __half, __half);
template void gelu<__half>(const Stream&, Span<__half>, View<__half>);
template void thresholdedrelu<__half>(const Stream&, Span<__half>, View<__half>, __half);
template void power<__half>(const Stream&, Span<__half>, View<__half>, __half, __half, __half);
template void exp<__half>(const Stream&, Span<__half>, View<__half>, __half, __half);
template void sign<__half>(const Stream&, Span<__half>, View<__half>);
template void shrink<__half>(const Stream&, Span<__half>, View<__half>, __half, __half);
template void reciprocal<__half>(const Stream&, Span<__half>, View<__half>);
#endif


template void relu<float>(const Stream&, Span<float>, View<float>, float);
template void clipped_relu<float>(const Stream&, Span<float>, View<float>, float, float);
template void tanh<float>(const Stream&, Span<float>, View<float>);
template void swish<float>(const Stream&, Span<float>, View<float>);
template void mish<float>(const Stream&, Span<float>, View<float>);
template void sigmoid<float>(const Stream&, Span<float>, View<float>);
template void elu<float>(const Stream&, Span<float>, View<float>, float);
template void abs<float>(const Stream& stream, Span<float> output, View<float> input);
template void bnll<float>(const Stream&, Span<float>, View<float>);
template void ceil<float>(const Stream&, Span<float>, View<float>);
template void floor<float>(const Stream&, Span<float>, View<float>);
template void log<float>(const Stream&, Span<float>, View<float>);
template void rint<float>(const Stream&, Span<float>, View<float>);
template void sqrt<float>(const Stream&, Span<float>, View<float>);
template void not_k<float>(const Stream&, Span<float>, View<float>);
template void acos<float>(const Stream&, Span<float>, View<float>);
template void acosh<float>(const Stream&, Span<float>, View<float>);
template void asin<float>(const Stream&, Span<float>, View<float>);
template void asinh<float>(const Stream&, Span<float>, View<float>);
template void atan<float>(const Stream&, Span<float>, View<float>);
template void atanh<float>(const Stream&, Span<float>, View<float>);
template void cos<float>(const Stream&, Span<float>, View<float>);
template void cosh<float>(const Stream&, Span<float>, View<float>);
template void erf<float>(const Stream&, Span<float>, View<float>);
template void hardswish<float>(const Stream&, Span<float>, View<float>);
template void sin<float>(const Stream&, Span<float>, View<float>);
template void sinh<float>(const Stream&, Span<float>, View<float>);
template void softplus<float>(const Stream&, Span<float>, View<float>);
template void softsign<float>(const Stream&, Span<float>, View<float>);
template void tan<float>(const Stream&, Span<float>, View<float>);
template void celu<float>(const Stream&, Span<float>, View<float>, float);
template void hardsigmoid<float>(const Stream&, Span<float>, View<float>, float, float);
template void selu<float>(const Stream&, Span<float>, View<float>, float, float);
template void gelu<float>(const Stream&, Span<float>, View<float>);
template void thresholdedrelu<float>(const Stream&, Span<float>, View<float>, float);
template void power<float>(const Stream&, Span<float>, View<float>, float, float, float);
template void exp<float>(const Stream&, Span<float>, View<float>, float, float);
template void sign<float>(const Stream&, Span<float>, View<float>);
template void shrink<float>(const Stream&, Span<float>, View<float>, float, float);
template void reciprocal<float>(const Stream&, Span<float>, View<float>);

template <class T, std::size_t N> static
void launch_vectorized_axiswise_relu(const Stream& stream, Span<T> output, View<T> input, std::size_t inner_size, View<T> slope) {
    CV_Assert(is_fully_aligned<T>(output, N));
    CV_Assert(is_fully_aligned<T>(input, N));
    CV_Assert(inner_size % N == 0);

    auto kernel = raw::axiswise_relu_vec<T, N>;
    auto policy = make_policy(kernel, output.size() / N, 0, stream);
    launch_kernel(kernel, policy, output, input, inner_size / N, slope);
}

template <class T>
void axiswise_relu(const Stream& stream, Span<T> output, View<T> input, std::size_t inner_size, View<T> slope) {
    CV_Assert(input.size() == output.size());

    if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(input, 4) && inner_size % 4 == 0) {
        launch_vectorized_axiswise_relu<T, 4>(stream, output, input, inner_size, slope);
    } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(input, 2) && inner_size % 2 == 0) {
        launch_vectorized_axiswise_relu<T, 2>(stream, output, input, inner_size, slope);
    } else {
        launch_vectorized_axiswise_relu<T, 1>(stream, output, input, inner_size, slope);
    }
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void axiswise_relu<__half>(const Stream&, Span<__half>, View<__half>, std::size_t, View<__half>);
#endif
    template void axiswise_relu<float>(const Stream&, Span<float>, View<float>, std::size_t, View<float>);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
