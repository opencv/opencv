#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "types.hpp"
#include "atomics.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

#include <cstddef>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

namespace raw {
    template <class T>
    __global__ void reduce_mean(Span<float> means, View<T> input, size_type inner_size) {
        for (auto idx : grid_stride_range(input.size())) {
            const index_type outer_idx = idx / inner_size;
            atomicAdd(&means[outer_idx], static_cast<float>(input[idx]) / inner_size);
        }
    }

    template <class T>
    __global__ void reduce_mean_sqr_sum(Span<float> means, Span<float> sum_sqrs, View<T> input, size_type inner_size) {
        for (auto idx : grid_stride_range(input.size())) {
            const index_type outer_idx = idx / inner_size;
            auto x = static_cast<float>(input[idx]);
            atomicAdd(&means[outer_idx], x / inner_size);
            atomicAdd(&sum_sqrs[outer_idx], x * x);
        }
    }

    __global__ void compute_normalization_scale(Span<float> scale, View<float> means, View<float> sums_sqr, size_type inner_size, float eps) {
        for (auto idx : grid_stride_range(scale.size())) {
            auto mean = means[idx];
            auto var = sums_sqr[idx] / inner_size - mean * mean;
            using device::rsqrt;
            scale[idx] = rsqrt(eps + var);
        }
    }

    template <class T>
    __global__ void normalize_mean(Span<T> output, View<T> input, View<float> means, size_type inner_size) {
        for (auto idx : grid_stride_range(output.size())) {
            const index_type outer_idx = idx / inner_size;
            output[idx] = static_cast<float>(input[idx]) - means[outer_idx];
        }
    }

    template <class T>
    __global__ void normalize_mean_variance(Span<T> output, View<T> input, View<float> means, View<float> scale, size_type inner_size) {
        for (auto idx : grid_stride_range(output.size())) {
            const index_type outer_idx = idx / inner_size;
            output[idx] = (static_cast<float>(input[idx]) - means[outer_idx]) * scale[outer_idx];
        }
    }

    template <class T>
    __global__ void normalize_mean_variance_channelwise(Span<T> output, View<T> input, View<T> scale, View<T> bias, View<float> means, View<float> inv_stddev, size_type inner_size, size_type C) {
        for (auto idx : grid_stride_range(output.size())) {
            const index_type outer_idx = idx / inner_size;
            const index_type c = outer_idx % C;
            auto s = static_cast<float>(scale[c]) * inv_stddev[outer_idx];
            auto b = static_cast<float>(bias[c]);
            output[idx] = (static_cast<float>(input[idx]) - means[outer_idx]) * s + b;
        }
    }

    template <class T>
    __global__ void normalize_mean_variance_groupwise(Span<T> output, View<T> input, View<T> scale, View<T> bias, View<float> means, View<float> inv_stddev, size_type inner_size, size_type C, size_type num_groups, size_type group_size) {
        for (auto idx : grid_stride_range(output.size())) {
            const index_type outer_idx = idx / inner_size;
            const index_type c = outer_idx % C;
            const index_type group_idx = outer_idx / group_size;
            auto s = static_cast<float>(scale[c]) * inv_stddev[group_idx];
            auto b = static_cast<float>(bias[c]);
            output[idx] = (static_cast<float>(input[idx]) - means[group_idx]) * s + b;
        }
    }

    template <class T>
    __global__ void normalize_mean_variance_layernorm(Span<T> output, View<T> input, View<T> scale, View<float> means, View<float> inv_stddev, size_type inner_size) {
        for (auto idx : grid_stride_range(output.size())) {
            const index_type outer_idx = idx / inner_size;
            const index_type inner_idx = idx % inner_size;
            auto s = static_cast<float>(scale[inner_idx]) * inv_stddev[outer_idx];
            output[idx] = (static_cast<float>(input[idx]) - means[outer_idx]) * s;
        }
    }

    template <class T>
    __global__ void normalize_mean_variance_layernorm_with_bias(Span<T> output, View<T> input, View<T> scale, View<T> bias, View<float> means, View<float> inv_stddev, size_type inner_size) {
        for (auto idx : grid_stride_range(output.size())) {
            const index_type outer_idx = idx / inner_size;
            const index_type inner_idx = idx % inner_size;
            auto s = static_cast<float>(scale[inner_idx]) * inv_stddev[outer_idx];
            auto b = static_cast<float>(bias[inner_idx]);
            output[idx] = (static_cast<float>(input[idx]) - means[outer_idx]) * s + b;
        }
    }
}

template <class T>
void reduce_mean(const Stream& stream, Span<float> means, View<T> input, std::size_t inner_size)
{
    CV_Assert(input.size() / inner_size == means.size());

    auto kernel = raw::reduce_mean<T>;
    auto policy = make_policy(kernel, input.size(), 0, stream);
    launch_kernel(kernel, policy, means, input, inner_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void reduce_mean(const Stream&, Span<float>, View<__half>, std::size_t);
#endif
template void reduce_mean(const Stream&, Span<float>, View<float>, std::size_t);

template <class T>
void reduce_mean_sqr_sum(const Stream& stream, Span<float> means, Span<float> sum_sqrs, View<T> input, std::size_t inner_size)
{
    CV_Assert(input.size() / inner_size == means.size());
    CV_Assert(input.size() / inner_size == sum_sqrs.size());

    auto kernel = raw::reduce_mean_sqr_sum<T>;
    auto policy = make_policy(kernel, input.size(), 0, stream);
    launch_kernel(kernel, policy, means, sum_sqrs, input, inner_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void reduce_mean_sqr_sum(const Stream&, Span<float>, Span<float>, View<__half>, std::size_t);
#endif
template void reduce_mean_sqr_sum(const Stream&, Span<float>, Span<float>, View<float>, std::size_t);

void compute_normalization_scale(const Stream& stream, Span<float> scale, View<float> means, View<float> sum_sqrs, std::size_t inner_size, float eps)
{
    CV_Assert(scale.size() == means.size());
    CV_Assert(scale.size() == sum_sqrs.size());

    auto kernel = raw::compute_normalization_scale;
    auto policy = make_policy(kernel, scale.size(), 0, stream);
    launch_kernel(kernel, policy, scale, means, sum_sqrs, inner_size, eps);
}

template <class T>
void normalize_mean(const Stream& stream, Span<T> output, View<T> input, View<float> means, std::size_t inner_size)
{
    CV_Assert(output.size() == input.size());
    CV_Assert(input.size() / inner_size == means.size());

    auto kernel = raw::normalize_mean<T>;
    auto policy = make_policy(kernel, output.size(), 0, stream);
    launch_kernel(kernel, policy, output, input, means, inner_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void normalize_mean(const Stream&, Span<__half>, View<__half>, View<float>, std::size_t);
#endif
template void normalize_mean(const Stream&, Span<float>, View<float>, View<float>, std::size_t);

template <class T>
void normalize_mean_variance(const Stream& stream, Span<T> output, View<T> input, View<float> means, View<float> scale, std::size_t inner_size)
{
    CV_Assert(input.size() == output.size());
    CV_Assert(input.size() / inner_size == means.size());
    CV_Assert(input.size() / inner_size == scale.size());

    auto kernel = raw::normalize_mean_variance<T>;
    auto policy = make_policy(kernel, output.size(), 0, stream);
    launch_kernel(kernel, policy, output, input, means, scale, inner_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void normalize_mean_variance(const Stream&, Span<__half>, View<__half>, View<float>, View<float>, std::size_t);
#endif
template void normalize_mean_variance(const Stream&, Span<float>, View<float>, View<float>, View<float>, std::size_t);

template <class T>
void normalize_mean_variance_channelwise(const Stream& stream, Span<T> output, View<T> input, View<T> scale, View<T> bias, View<float> means, View<float> inv_stddev, std::size_t inner_size, std::size_t C)
{
    CV_Assert(input.size() == output.size());
    CV_Assert(input.size() / inner_size == means.size());
    CV_Assert(means.size() == inv_stddev.size());

    auto kernel = raw::normalize_mean_variance_channelwise<T>;
    auto policy = make_policy(kernel, output.size(), 0, stream);
    launch_kernel(kernel, policy, output, input, scale, bias, means, inv_stddev, inner_size, C);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void normalize_mean_variance_channelwise(const Stream&, Span<__half> /*output*/, View<__half> /*input*/, View<__half> /*scale*/, View<__half> /*bias*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t, std::size_t);
#endif
template void normalize_mean_variance_channelwise(const Stream&, Span<float> /*output*/, View<float> /*input*/, View<float> /*scale*/, View<float> /*bias*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t, std::size_t);

template <class T>
void normalize_mean_variance_groupwise(const Stream& stream, Span<T> output, View<T> input, View<T> scale, View<T> bias, View<float> means, View<float> inv_stddev, std::size_t inner_size, std::size_t C, std::size_t num_groups, std::size_t group_size)
{
    CV_Assert(input.size() == output.size());
    CV_Assert(input.size() / inner_size == means.size() * group_size);
    CV_Assert(means.size() == inv_stddev.size());

    auto kernel = raw::normalize_mean_variance_groupwise<T>;
    auto policy = make_policy(kernel, output.size(), 0, stream);
    launch_kernel(kernel, policy, output, input, scale, bias, means, inv_stddev, inner_size, C, num_groups, group_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void normalize_mean_variance_groupwise(const Stream&, Span<__half> /*output*/, View<__half> /*input*/, View<__half> /*scale*/, View<__half> /*bias*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t, std::size_t, std::size_t, std::size_t);
#endif
template void normalize_mean_variance_groupwise(const Stream&, Span<float> /*output*/, View<float> /*input*/, View<float> /*scale*/, View<float> /*bias*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t, std::size_t, std::size_t, std::size_t);


template <class T>
void normalize_mean_variance_layernorm(const Stream& stream, Span<T> output, View<T> input, View<T> scale, View<float> means, View<float> inv_stddev, std::size_t inner_size)
{
    CV_Assert(input.size() == output.size());
    CV_Assert(input.size() / inner_size == means.size());
    CV_Assert(means.size() == inv_stddev.size());

    auto kernel = raw::normalize_mean_variance_layernorm<T>;
    auto policy = make_policy(kernel, output.size(), 0, stream);
    launch_kernel(kernel, policy, output, input, scale, means, inv_stddev, inner_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void normalize_mean_variance_layernorm(const Stream&, Span<__half> /*output*/, View<__half> /*input*/, View<__half> /*scale*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t);
#endif
template void normalize_mean_variance_layernorm(const Stream&, Span<float> /*output*/, View<float> /*input*/, View<float> /*scale*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t);

template <class T>
void normalize_mean_variance_layernorm(const Stream& stream, Span<T> output, View<T> input, View<T> scale, View<T> bias, View<float> means, View<float> inv_stddev, std::size_t inner_size)
{
    CV_Assert(input.size() == output.size());
    CV_Assert(input.size() / inner_size == means.size());
    CV_Assert(means.size() == inv_stddev.size());

    auto kernel = raw::normalize_mean_variance_layernorm_with_bias<T>;
    auto policy = make_policy(kernel, output.size(), 0, stream);
    launch_kernel(kernel, policy, output, input, scale, bias, means, inv_stddev, inner_size);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void normalize_mean_variance_layernorm(const Stream&, Span<__half> /*output*/, View<__half> /*input*/, View<__half> /*scale*/, View<__half> /*bias*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t);
#endif
template void normalize_mean_variance_layernorm(const Stream&, Span<float> /*output*/, View<float> /*input*/, View<float> /*scale*/, View<float> /*bias*/, View<float> /*means*/, View<float> /*inv_stddev*/, std::size_t);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
