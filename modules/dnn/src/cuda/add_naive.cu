#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>
#include "conv_naive.hpp"
#include <hipDNN.h>
#include <cstdio>

namespace cv { namespace dnn { namespace cuda_naive_conv {

void add2_fp32_2d(
    const float* d_a, size_t a_step,
    const float* d_b, size_t b_step,
    float* d_y, size_t y_step,
    int rows, int cols)
{
    {
        hipdnnHandle_t handle = nullptr;
        hipdnnTensorDescriptor_t aDesc = nullptr, bDesc = nullptr, yDesc = nullptr;
        hipdnnOpTensorDescriptor_t opDesc = nullptr;
        hipdnnStatus_t st = HIPDNN_STATUS_SUCCESS; bool ok = true;
        if ((st = hipdnnCreate(&handle)) != HIPDNN_STATUS_SUCCESS) return;
        hipdnnSetStream(handle, 0);
        if (ok && (st = hipdnnCreateTensorDescriptor(&aDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateTensorDescriptor(&bDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateTensorDescriptor(&yDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateOpTensorDescriptor(&opDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;

        int n = rows, c = 1, h = 1, w = cols;
        int a_stride_w = 1;
        int a_stride_h = w;
        int a_stride_c = h * w;
        int a_stride_n = (int)(a_step / sizeof(float));
        int b_stride_w = 1;
        int b_stride_h = w;
        int b_stride_c = h * w;
        int b_stride_n = (int)(b_step / sizeof(float));
        int y_stride_w = 1;
        int y_stride_h = w;
        int y_stride_c = h * w;
        int y_stride_n = (int)(y_step / sizeof(float));
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(aDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          a_stride_n, a_stride_c, a_stride_h, a_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(bDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          b_stride_n, b_stride_c, b_stride_h, b_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(yDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          y_stride_n, y_stride_c, y_stride_h, y_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;

        if (ok && (st = hipdnnSetOpTensorDescriptor(opDesc, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN)) != HIPDNN_STATUS_SUCCESS) ok = false;

        const float alpha1 = 1.0f, alpha2 = 1.0f, beta = 0.0f;
        if (ok && (st = hipdnnOpTensor(handle, opDesc,
                          &alpha1, aDesc, d_a,
                          &alpha2, bDesc, d_b,
                          &beta, yDesc, d_y)) != HIPDNN_STATUS_SUCCESS) ok = false;

        if (ok) std::fprintf(stderr, "DNN(cuDNN): add2_fp32_2d using cuDNN (rows=%d cols=%d)\n", rows, cols);
        else std::fprintf(stderr, "DNN(cuDNN): add2_fp32_2d cuDNN error: %s\n", hipdnnGetErrorString(st));

        if (opDesc) hipdnnDestroyOpTensorDescriptor(opDesc);
        if (yDesc) hipdnnDestroyTensorDescriptor(yDesc);
        if (bDesc) hipdnnDestroyTensorDescriptor(bDesc);
        if (aDesc) hipdnnDestroyTensorDescriptor(aDesc);
        if (handle) hipdnnDestroy(handle);
        return;
    }
}

static __global__ void add_inplace_fp32_2d_kernel(
    const float* __restrict__ x, size_t x_step,
    float* __restrict__ y, size_t y_step,
    int rows, int cols)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < rows && c < cols) {
        const float* xr = (const float*)((const unsigned char*)x + r * x_step);
        float* yr = (float*)((unsigned char*)y + r * y_step);
        yr[c] = yr[c] + xr[c];
    }
}

void add_inplace_fp32_2d(
    const float* d_x, size_t x_step,
    float* d_y, size_t y_step,
    int rows, int cols)
{
    {
        hipdnnHandle_t handle = nullptr;
        hipdnnTensorDescriptor_t xDesc = nullptr, yDesc = nullptr;
        hipdnnOpTensorDescriptor_t opDesc = nullptr;
        hipdnnStatus_t st = HIPDNN_STATUS_SUCCESS; bool ok = true;
        if ((st = hipdnnCreate(&handle)) != HIPDNN_STATUS_SUCCESS) return;
        hipdnnSetStream(handle, 0);
        if (ok && (st = hipdnnCreateTensorDescriptor(&xDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateTensorDescriptor(&yDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateOpTensorDescriptor(&opDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;

        int n = rows, c = 1, h = 1, w = cols;
        int x_stride_w = 1;
        int x_stride_h = w;
        int x_stride_c = h * w;
        int x_stride_n = (int)(x_step / sizeof(float));
        int y_stride_w = 1;
        int y_stride_h = w;
        int y_stride_c = h * w;
        int y_stride_n = (int)(y_step / sizeof(float));
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(xDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          x_stride_n, x_stride_c, x_stride_h, x_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(yDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          y_stride_n, y_stride_c, y_stride_h, y_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnSetOpTensorDescriptor(opDesc, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN)) != HIPDNN_STATUS_SUCCESS) ok = false;

        const float alpha1 = 1.0f, alpha2 = 1.0f, beta = 1.0f;
        if (ok && (st = hipdnnOpTensor(handle, opDesc,
                          &alpha1, yDesc, d_y,
                          &alpha2, xDesc, d_x,
                          &beta, yDesc, d_y)) != HIPDNN_STATUS_SUCCESS) ok = false;

        if (ok) std::fprintf(stderr, "DNN(cuDNN): add_inplace_fp32_2d using cuDNN (rows=%d cols=%d)\n", rows, cols);
        else std::fprintf(stderr, "DNN(cuDNN): add_inplace_fp32_2d cuDNN error: %s\n", hipdnnGetErrorString(st));

        if (opDesc) hipdnnDestroyOpTensorDescriptor(opDesc);
        if (yDesc) hipdnnDestroyTensorDescriptor(yDesc);
        if (xDesc) hipdnnDestroyTensorDescriptor(xDesc);
        if (handle) hipdnnDestroy(handle);
        return;
    }
}

}}} // namespace cv::dnn::cuda_naive_conv
