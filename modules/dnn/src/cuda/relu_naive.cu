#include <hip/hip_runtime.h>
#include <float.h>
#include "conv_naive.hpp"
#include <hipDNN.h>
#include <cstdio>

namespace cv { namespace dnn { namespace cuda_naive_conv {

void relu_fp32(const float* d_input, float* d_output, size_t count)
{
    {
        if (count == 0) { return; }
        hipdnnHandle_t handle = nullptr;
        hipdnnTensorDescriptor_t xDesc = nullptr, yDesc = nullptr;
        hipdnnActivationDescriptor_t actDesc = nullptr;
        hipdnnStatus_t st = HIPDNN_STATUS_SUCCESS; bool ok = true;
        if ((st = hipdnnCreate(&handle)) != HIPDNN_STATUS_SUCCESS) return;
        hipdnnSetStream(handle, 0);
        if (ok && (st = hipdnnCreateTensorDescriptor(&xDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateTensorDescriptor(&yDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateActivationDescriptor(&actDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;

        int n = 1, c = (int)count, h = 1, w = 1;
        int stride_w = 1, stride_h = 1, stride_c = 1, stride_n = c;
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(xDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          stride_n, stride_c, stride_h, stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(yDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          stride_n, stride_c, stride_h, stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0)) != HIPDNN_STATUS_SUCCESS) ok = false;

        const float alpha = 1.0f, beta = 0.0f;
        if (ok && (st = hipdnnActivationForward(handle, actDesc,
                                   &alpha, xDesc, d_input,
                                   &beta, yDesc, d_output)) != HIPDNN_STATUS_SUCCESS) ok = false;

        if (ok) std::fprintf(stderr, "DNN(cuDNN): relu_fp32 using cuDNN (count=%zu)\n", count);
        else std::fprintf(stderr, "DNN(cuDNN): relu_fp32 cuDNN error: %s\n", hipdnnGetErrorString(st));

        if (actDesc) hipdnnDestroyActivationDescriptor(actDesc);
        if (yDesc) hipdnnDestroyTensorDescriptor(yDesc);
        if (xDesc) hipdnnDestroyTensorDescriptor(xDesc);
        if (handle) hipdnnDestroy(handle);
        return;
    }
}

// removed kernel; cuDNN-only implementation

void relu_fp32_2d(const float* d_input, size_t input_step, float* d_output, size_t output_step, int rows, int cols)
{
    {
        hipdnnHandle_t handle = nullptr;
        hipdnnTensorDescriptor_t xDesc = nullptr, yDesc = nullptr;
        hipdnnActivationDescriptor_t actDesc = nullptr;
        hipdnnStatus_t st = HIPDNN_STATUS_SUCCESS; bool ok = true;
        if ((st = hipdnnCreate(&handle)) != HIPDNN_STATUS_SUCCESS) return;
        hipdnnSetStream(handle, 0);
        if (ok && (st = hipdnnCreateTensorDescriptor(&xDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateTensorDescriptor(&yDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;
        if (ok && (st = hipdnnCreateActivationDescriptor(&actDesc)) != HIPDNN_STATUS_SUCCESS) ok = false;

        int n = rows, c = cols, h = 1, w = 1;
        int x_stride_w = 1;
        int x_stride_h = 1;
        int x_stride_c = 1;
        int x_stride_n = (int)(input_step / sizeof(float));
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(xDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          x_stride_n, x_stride_c, x_stride_h, x_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;

        int y_stride_w = 1;
        int y_stride_h = 1;
        int y_stride_c = 1;
        int y_stride_n = (int)(output_step / sizeof(float));
        if (ok && (st = hipdnnSetTensor4dDescriptorEx(yDesc, HIPDNN_DATA_FLOAT, n, c, h, w,
                                          y_stride_n, y_stride_c, y_stride_h, y_stride_w)) != HIPDNN_STATUS_SUCCESS) ok = false;

        if (ok && (st = hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0)) != HIPDNN_STATUS_SUCCESS) ok = false;

        const float alpha = 1.0f, beta = 0.0f;
        if (ok && (st = hipdnnActivationForward(handle, actDesc,
                                   &alpha, xDesc, d_input,
                                   &beta, yDesc, d_output)) != HIPDNN_STATUS_SUCCESS) ok = false;

        if (ok) std::fprintf(stderr, "DNN(cuDNN): relu_fp32_2d using cuDNN (rows=%d cols=%d)\n", rows, cols);
        else std::fprintf(stderr, "DNN(cuDNN): relu_fp32_2d cuDNN error: %s\n", hipdnnGetErrorString(st));

        if (actDesc) hipdnnDestroyActivationDescriptor(actDesc);
        if (yDesc) hipdnnDestroyTensorDescriptor(yDesc);
        if (xDesc) hipdnnDestroyTensorDescriptor(xDesc);
        if (handle) hipdnnDestroy(handle);
        return;
    }
}

}}} // namespace cv::dnn::cuda_naive_conv
